#include "../inc/memory.h"

inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cout << hipGetErrorString(code) << std::endl;
        exit(-1);
    }
}

void malloc_graph_to_gpu_memory(Graph &g, G_pointers &p, bool query)
{

    // std::cout<<"1"<<std::endl;
    chkerr(hipMalloc(&(p.outgoing_neighbors), g.outgoing_neighbors_offset[g.V] * sizeof(unsigned int)));
    chkerr(hipMemcpy(p.outgoing_neighbors, g.outgoing_neighbors, g.outgoing_neighbors_offset[g.V] * sizeof(unsigned int), hipMemcpyHostToDevice));

    // std::cout<<"2"<<std::endl;
    chkerr(hipMalloc(&(p.outgoing_neighbors_offset), (g.V + 1) * sizeof(unsigned int)));
    chkerr(hipMemcpy(p.outgoing_neighbors_offset, g.outgoing_neighbors_offset, (g.V + 1) * sizeof(unsigned int), hipMemcpyHostToDevice));

    // signatures
    // std::cout<<"3"<<std::endl;
    chkerr(hipMalloc(&(p.signatures), (g.V) * sizeof(unsigned int) * Signature_Properties));
    chkerr(hipMemcpy(p.signatures, g.signatures, (g.V) * sizeof(unsigned int) * Signature_Properties, hipMemcpyHostToDevice));

    // incoming neighbors now
    // std::cout<<"4"<<std::endl;
    chkerr(hipMalloc(&(p.incoming_neighbors), g.incoming_neighbors_offset[g.V] * sizeof(unsigned int)));
    chkerr(hipMemcpy(p.incoming_neighbors, g.incoming_neighbors, g.incoming_neighbors_offset[g.V] * sizeof(unsigned int), hipMemcpyHostToDevice));

    // std::cout<<"5"<<std::endl;
    chkerr(hipMalloc(&(p.incoming_neighbors_offset), (g.V + 1) * sizeof(unsigned int)));
    chkerr(hipMemcpy(p.incoming_neighbors_offset, g.incoming_neighbors_offset, (g.V + 1) * sizeof(unsigned int), hipMemcpyHostToDevice));

    // attributes
    // std::cout<<"6"<<std::endl;
    chkerr(hipMalloc(&(p.attributes), (g.V) * sizeof(unsigned int)));
    chkerr(hipMemcpy(p.attributes, g.attributes, (g.V) * sizeof(unsigned int), hipMemcpyHostToDevice));

    // std::cout<<"7"<<std::endl;
    chkerr(hipMalloc(&(p.attributes_in_order), g.attributes_in_order_offset[g.num_attributes] * sizeof(unsigned int)));
    chkerr(hipMemcpy(p.attributes_in_order, g.attributes_in_order, g.attributes_in_order_offset[g.num_attributes] * sizeof(unsigned int), hipMemcpyHostToDevice));

    // std::cout<<"8"<<std::endl;
    chkerr(hipMalloc(&(p.attributes_in_order_offset), (g.num_attributes + 1) * sizeof(unsigned int)));
    chkerr(hipMemcpy(p.attributes_in_order_offset, g.attributes_in_order_offset, (g.num_attributes + 1) * sizeof(unsigned int), hipMemcpyHostToDevice));

    std::cout << "7" << std::endl;
    chkerr(hipMalloc(&(p.all_neighbors), g.all_neighbors_offset[g.V] * sizeof(unsigned int)));
    chkerr(hipMemcpy(p.all_neighbors, g.all_neighbors, g.all_neighbors_offset[g.V] * sizeof(unsigned int), hipMemcpyHostToDevice));

    std::cout << "8" << std::endl;
    chkerr(hipMalloc(&(p.all_neighbors_offset), (g.V + 1) * sizeof(unsigned int)));
    chkerr(hipMemcpy(p.all_neighbors_offset, g.all_neighbors_offset, (g.V + 1) * sizeof(unsigned int), hipMemcpyHostToDevice));

    std::cout << "9" << std::endl;
    p.V = g.V;

    p.E = g.E;

    // std::cout<<"10"<<std::endl;
    p.num_attributes = g.num_attributes;
}

void malloc_extra_to_gpu_memory(E_pointers &e, unsigned int v, unsigned int *v_order)
{

    // so cudaMemoryManaged allows cpu and gpu have access to memory at a cost (speed)
    // need this for our results table

    chkerr(hipMalloc(&(e.matching_order), (v) * sizeof(unsigned int)));
    chkerr(hipMemcpy(e.matching_order, v_order, (v) * sizeof(unsigned int), hipMemcpyHostToDevice));

    chkerr(hipMalloc(&(e.global_count), sizeof(unsigned int)));
    chkerr(hipMemset(e.global_count, 0, sizeof(unsigned int)));

    chkerr(hipMalloc(&(e.global_count), sizeof(unsigned int)));
    chkerr(hipMemset(e.global_count, 0, sizeof(unsigned int)));

    chkerr(hipMallocManaged(&(e.result_lengths), (v + 2) * sizeof(unsigned int)));

    // datastructure
    // this table_size is variable based on workers, but for now im setting it constant
    unsigned int table_size = GPU_TABLE_SIZES;
    chkerr(hipMalloc(&(e.results_table), table_size));
    chkerr(hipMalloc(&(e.indexes_table), table_size));
    chkerr(hipMalloc(&(e.scores_table), table_size));
    chkerr(hipMalloc(&(e.intra_v_table), table_size));

    chkerr(hipMallocManaged(&(e.write_pos), sizeof(unsigned long long int)));
    chkerr(hipMemset(e.write_pos, 0, sizeof(unsigned long long int)));

    unsigned int buffer_size = BUFFER_TABLE_SIZE * sizeof(unsigned int);
    chkerr(hipMalloc(&(e.helper_buffer), buffer_size));

    
}