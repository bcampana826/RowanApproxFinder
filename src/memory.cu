#include "../inc/memory.h"


inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cout<<hipGetErrorString(code)<<std::endl;
        exit(-1);
    }
}

void malloc_graph_to_gpu_memory(Graph &g, G_pointers &p, bool query){


    //std::cout<<"1"<<std::endl;
    chkerr(hipMalloc(&(p.outgoing_neighbors),g.outgoing_neighbors_offset[g.V]*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.outgoing_neighbors,g.outgoing_neighbors,g.outgoing_neighbors_offset[g.V]*sizeof(unsigned int),hipMemcpyHostToDevice));
   
    //std::cout<<"2"<<std::endl;
    chkerr(hipMalloc(&(p.outgoing_neighbors_offset),(g.V+1)*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.outgoing_neighbors_offset,g.outgoing_neighbors_offset,(g.V+1)*sizeof(unsigned int),hipMemcpyHostToDevice));

    // signatures
    //std::cout<<"3"<<std::endl;
    chkerr(hipMalloc(&(p.signatures),(g.V)*sizeof(unsigned int)*Signature_Properties));
    chkerr(hipMemcpy(p.signatures,g.signatures,(g.V)*sizeof(unsigned int)*Signature_Properties,hipMemcpyHostToDevice));

    // incoming neighbors now
    //std::cout<<"4"<<std::endl;
    chkerr(hipMalloc(&(p.incoming_neighbors),g.incoming_neighbors_offset[g.V]*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.incoming_neighbors,g.incoming_neighbors,g.incoming_neighbors_offset[g.V]*sizeof(unsigned int),hipMemcpyHostToDevice));

    //std::cout<<"5"<<std::endl;
    chkerr(hipMalloc(&(p.incoming_neighbors_offset),(g.V+1)*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.incoming_neighbors_offset,g.incoming_neighbors_offset,(g.V+1)*sizeof(unsigned int),hipMemcpyHostToDevice));

    // attributes
    //std::cout<<"6"<<std::endl;
    chkerr(hipMalloc(&(p.attributes),(g.V)*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.attributes,g.attributes,(g.V)*sizeof(unsigned int),hipMemcpyHostToDevice));

    //std::cout<<"7"<<std::endl;
    chkerr(hipMalloc(&(p.attributes_in_order),g.attributes_in_order_offset[g.largest_att+1]*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.attributes_in_order,g.attributes_in_order,g.attributes_in_order_offset[g.largest_att+1]*sizeof(unsigned int),hipMemcpyHostToDevice));

    //std::cout<<"8"<<std::endl;
    chkerr(hipMalloc(&(p.attributes_in_order_offset),(g.largest_att+1+1)*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.attributes_in_order_offset,g.attributes_in_order_offset,(g.largest_att+1+1)*sizeof(unsigned int),hipMemcpyHostToDevice));

    //std::cout<<"9"<<std::endl;
    p.V = g.V;

    p.E = g.E;

    //std::cout<<"10"<<std::endl;
    p.largest_att = g.largest_att;

}

void malloc_extra_to_gpu_memory(E_pointers &e, unsigned int v, unsigned int *v_order){

    // so cudaMemoryManaged allows cpu and gpu have access to memory at a cost (speed)
    // need this for our results table

    chkerr(hipMalloc(&(e.matching_order),(v)*sizeof(unsigned int)));
    chkerr(hipMemcpy(e.matching_order,v_order,(v)*sizeof(unsigned int),hipMemcpyHostToDevice));

    chkerr(hipMalloc(&(e.result_lengths),(v)*sizeof(unsigned int)));

    // datastructure
    // this table_size is variable based on workers, but for now im setting it constant
    unsigned int table_size = 1 * sizeof(unsigned int);
    chkerr(hipMalloc(&(e.results_table),table_size));
    chkerr(hipMalloc(&(e.indexes_table),table_size));
    chkerr(hipMalloc(&(e.scores_table),table_size));
    chkerr(hipMalloc(&(e.intra_v_table),table_size));


    chkerr(hipMalloc(&(e.write_pos),sizeof(unsigned long long int)));
    chkerr(hipMemset(e.write_pos,0,sizeof(unsigned long long int)));

}