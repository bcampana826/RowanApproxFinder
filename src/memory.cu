#include "../inc/memory.h"


inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cout<<hipGetErrorString(code)<<std::endl;
        exit(-1);
    }
}

void malloc_graph_to_gpu_memory(Graph &g, G_pointers &p, bool query){

    chkerr(hipMalloc(&(p.outgoing_neighbors),g.outgoing_neighbors_offset[g.V]*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.outgoing_neighbors,g.outgoing_neighbors,g.outgoing_neighbors_offset[g.V]*sizeof(unsigned int),hipMemcpyHostToDevice));
   
    chkerr(hipMalloc(&(p.outgoing_neighbors_offset),(g.V+1)*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.outgoing_neighbors_offset,g.outgoing_neighbors_offset,(g.V+1)*sizeof(unsigned int),hipMemcpyHostToDevice));

    // signatures
    chkerr(hipMalloc(&(p.signatures),(g.V)*sizeof(unsigned int)*Signature_Properties));
    chkerr(hipMemcpy(p.signatures,g.signatures,(g.V)*sizeof(unsigned int)*Signature_Properties,hipMemcpyHostToDevice));

    // incoming neighbors now
    chkerr(hipMalloc(&(p.incoming_neighbors),g.incoming_neighbors_offset[g.V]*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.incoming_neighbors,g.incoming_neighbors,g.incoming_neighbors_offset[g.V]*sizeof(unsigned int),hipMemcpyHostToDevice));

    chkerr(hipMalloc(&(p.incoming_neighbors_offset),(g.V+1)*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.incoming_neighbors_offset,g.incoming_neighbors_offset,(g.V+1)*sizeof(unsigned int),hipMemcpyHostToDevice));

    // attributes
    chkerr(hipMalloc(&(p.attributes),(g.V)*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.attributes,g.attributes,(g.V)*sizeof(unsigned int),hipMemcpyHostToDevice));

    chkerr(hipMalloc(&(p.attributes_in_order),g.attributes_in_order_offset[g.largest_att+1]*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.attributes_in_order,g.attributes_in_order,g.attributes_in_order_offset[g.largest_att+1]*sizeof(unsigned int),hipMemcpyHostToDevice));

    chkerr(hipMalloc(&(p.attributes_in_order_offset),(g.V+1)*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.attributes_in_order_offset,g.attributes_in_order_offset,(g.V+1)*sizeof(unsigned int),hipMemcpyHostToDevice));


    p.V = g.V;

    p.largest_att = g.largest_att;

}